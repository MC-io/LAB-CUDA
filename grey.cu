#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>

__global__ void grey_scale(int * img,int * grey_img, int height, int width) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int index = i * (width * 3) + (j * 3);


    if (i > height) return;
    if (j > width) return;
    grey_img[i * width + j] = ((float)img[index] * 0.21f) + ((float)img[index + 1] * 0.72f) + ((float)img[index + 2] * 0.07f);
}

int main() {

    std::cout << "Hola\n";

    std::ifstream file;
    file.open("C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\grey_scale\\image.txt");
    int width = 10, height = 10;
    file >> height >> width;

    int* img = (int*)malloc(height * width * 3 * sizeof(int));
    int* grey_img = (int*)malloc(height * width * sizeof(int));

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            int index = i * (width * 3) + (j * 3);
            file >> img[index] >> img[index + 1] >> img[index + 2];
        }
    }

    file.close();

    dim3 threadsPerBlock(16, 16);

    int bx = std::ceil((float)height / 16.f);
    int by = std::ceil((float)width / 16.f);


    dim3 numBlocks(bx, by);

    int* d_img, * d_grey_img;

    hipMalloc(&d_img, height * width * 3 * sizeof(int));
    hipMalloc(&d_grey_img, height * width * sizeof(int));
    
    hipMemcpy(d_img, img, (height * width * 3) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_grey_img, grey_img, (height * width) * sizeof(int), hipMemcpyHostToDevice);

    
    grey_scale << <numBlocks, threadsPerBlock >> > (d_img, d_grey_img, height, width);
    
    hipMemcpy(grey_img, d_grey_img, (height * width) * sizeof(int), hipMemcpyDeviceToHost);

    
    std::ofstream res;
    res.open("C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\grey_scale\\new_image.txt");


    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            res << grey_img[i * width + j] << '\n';
        }
    }

    res.close();

    //system("python C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\write_image.py");
    
    free(img);
    free(grey_img);

    hipFree(d_img);
    hipFree(d_grey_img);

    printf("QUE");

    return 0;
}