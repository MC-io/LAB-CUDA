﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>

#define HEIGHT 64
#define WIDTH 64

__global__ void blur_image(int * img, int * blurred_img, int height, int width) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > height) return;
    if (j > width) return;
    

    int index = i * (width * 3) + (j * 3);
    blurred_img[index + 0] = 0;
    blurred_img[index + 1] = 0;
    blurred_img[index + 2] = 0;

    int count = 0;
    for (int ii = i - 1; ii < i + 2; ii++)
    {
        if (ii < 0) continue;
        else if (ii >= height) break;
        for (int jj = j - 1; jj < j + 2; jj++)
        {
            if (jj < 0) continue;
            else if (jj >= width) break;
            int ind_n = ii * (width * 3) + (jj * 3);
            blurred_img[index + 0] += img[ind_n + 0];
            blurred_img[index + 1] += img[ind_n + 1];
            blurred_img[index + 2] += img[ind_n + 2];
            count++;
        }
    }

    blurred_img[index + 0] /= count;
    blurred_img[index + 1] /= count;
    blurred_img[index + 2] /= count;
}


int main() {

    std::cout << "Hola\n";
    
    std::ifstream file;
    file.open("C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\image.txt");
    int width= 10, height = 10;
    file >> height >> width;

    int * img = (int*)malloc(height * width * 3 * sizeof(int));
    int * blurred_img = (int*)malloc(height * width * 3 * sizeof(int));
    
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            int index = i * (width * 3) + (j * 3);
            file >> img[index] >> img[index + 1] >> img[index + 2];
        }
    }
    
    file.close();
  
    dim3 threadsPerBlock(16, 16);

    int bx = std::ceil((float)height / 16.f);
    int by = std::ceil((float)width / 16.f);


    dim3 numBlocks(bx, by);
    
    int* d_img, * d_blurred_img;
    
    hipMalloc(&d_img, height * width * 3 * sizeof(int));
    hipMalloc(&d_blurred_img, height * width * 3 * sizeof(int));
    
    hipMemcpy(d_img, img, (height * width * 3) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blurred_img, blurred_img, (height * width * 3) * sizeof(int), hipMemcpyHostToDevice);
    

    blur_image <<<numBlocks,threadsPerBlock>>> (d_img, d_blurred_img, height, width);
    
    hipMemcpy(blurred_img, d_blurred_img, (height * width * 3) * sizeof(int), hipMemcpyDeviceToHost);
    

    std::ofstream res;
    res.open("C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\new_image.txt");
    

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            res << blurred_img[i * (width * 3) + (j * 3)] << '\n' << blurred_img[i * (width * 3) + (j * 3) + 1] << '\n' << blurred_img[i * (width * 3) + (j * 3) + 2] << '\n';
        }
    }

    res.close();

    //system("python C:\\Users\\pc\\Desktop\\8vo Semestre\\Computacion Paralela y Distribuida\\Lab cuda\\write_image.py");
   
    free(img);
    free(blurred_img);
   
    hipFree(d_img);
    hipFree(d_blurred_img);    

    printf("QUE");
    
    return 0;
}